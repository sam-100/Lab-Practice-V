
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

// CUDA Kernel to perform matrix multiplication
__global__ void matrixMultiplication(int *a, int *b, int *c, int m, int n, int p) {
    // Calculate the row and column index of the element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < p) {
        int sum = 0;
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i * p + col];
        }
        c[row * p + col] = sum;
    }
}

int main() {
    int m, n, p;

    // Input matrix dimensions
    std::cout << "Enter number of rows (m) for matrix A: ";
    std::cin >> m;
    std::cout << "Enter number of columns (n) for matrix A and rows for matrix B: ";
    std::cin >> n;
    std::cout << "Enter number of columns (p) for matrix B: ";
    std::cin >> p;

    // Allocate memory for matrices A, B, and C on host
    int *h_A = new int[m * n];
    int *h_B = new int[n * p];
    int *h_C = new int[m * p];

    // Initialize matrices A and B with random values
    srand(time(NULL));
    for (int i = 0; i < m * n; ++i) {
        h_A[i] = rand() % 10;  // Random numbers between 0 and 9
    }
    for (int i = 0; i < n * p; ++i) {
        h_B[i] = rand() % 10;  // Random numbers between 0 and 9
    }

    // Allocate memory for matrices A, B, and C on device
    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * n * sizeof(int));
    hipMalloc((void**)&d_B, n * p * sizeof(int));
    hipMalloc((void**)&d_C, m * p * sizeof(int));

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, h_A, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * p * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((p + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel to perform matrix multiplication
    matrixMultiplication<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, m, n, p);

    // Copy matrix C from device to host
    hipMemcpy(h_C, d_C, m * p * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Output the resulting matrix C
    std::cout << "Resultant matrix C (" << m << "x" << p << "):" << std::endl;
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < p; ++j) {
            std::cout << h_C[i * p + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
